
#include <hip/hip_runtime.h>
#include <stdio.h>

// Size of array
#define N 100000

// Kernel
__global__ void add_vectors(double *a, double *b, double *c)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < N) c[id] = a[id] + b[id];
}

// Main program
int main()
{
    // Number of bytes to allocate for N doubles
    size_t bytes = N*sizeof(double);

    // Allocate memory for arrays A, B, and C on host
    double *A = (double*)malloc(bytes);
    double *B = (double*)malloc(bytes);
    double *C = (double*)malloc(bytes);

    // Allocate memory for arrays d_A, d_B, and d_C on device
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Fill host arrays A and B
    for(int i=0; i<N; i++)
    {
        A[i] = 1.0;
        B[i] = 2.0;
    }

    // Copy data from host arrays A and B to device arrays d_A and d_B
    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    // Set execution configuration parameters
    //      thr_per_blk: number of CUDA threads per grid block
    //      blk_in_grid: number of blocks in grid
    int thr_per_blk = 256;
    int blk_in_grid = 64;

    // Launch kernel
    add_vectors<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_C);

    // Copy data from device array d_C to host array C
    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\n---------------------------\n");
    printf("__SUCCESS__\n");
    printf("---------------------------\n");
    printf("N = %d\n", N);
    printf("A = %f\n", A[0]);
    printf("B = %f\n", B[0]);
    printf("C = %f\n", C[0]);
    printf("---------------------------\n\n");

    //Free CPU memory
    free(A);
    free(B);
    free(C);

    return 0;
}
