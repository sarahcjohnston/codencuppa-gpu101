#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "timer.h"

//1. Define a set of initial particles
//2. Calculate the forces on each of those particles
//3. Update the particles with how the force on them affects their position and velocity
//4. Repeat with the new set of particles


//This is the kernel (where the calculation happens)
__global__ void force(float *x, float *y, float *z, float *vx, float *vy, float *vz, float *m, float dt, int n, double G){	  
	   int t = blockIdx.x*blockDim.x +threadIdx.x;
	   int T = blockDim.x*gridDim.x;

	   //create loop to calculate forces
	   for(int i = t; i < n; i+=T){
	   	//initialise forces to 0
	      float Fx = 0.0;
	      float Fy = 0.0;
	      float Fz = 0.0;

	      //find forces for each of the n bodies
	      for(int j =0; j<n; j++){
	      		//skip forces between particle and itself
		      if(j==i){
		      		continue;}
	      	      //calculate distance between particle and neighbours for each of the particles
	      	      float dx = x[j] - x[i]; 
		      float dy = y[j] - y[i];
		      float dz = z[j] - z[i];

		      //find total distance by squaring
		      float totdist = rsqrt(dx*dx + dy*dy + dz*dz);
		      float distcube = totdist*totdist*totdist;

		      //find the forces and add them on to the initial value
		      Fx += G*m[j]*dx*distcube;
		      Fy += G*m[j]*dy*distcube;
		      Fz += G*m[j]*dz*distcube;
		      }
		      
	       //update the velocity values from the new forces
	       vx[i] += dt*Fx;
	       vy[i] += dt*Fy;
	       vz[i] += dt*Fz;

		//update position values from the new forces
	       x[i] += vx[i]*dt;
	       y[i] += vy[i]*dt;
	       z[i] += vz[i]*dt;
	       }

}

int main(int argc, char *argv[]){

    //make file for snapshots
    FILE *fptr1, *fptr2, *fptr3;

	//USER-INPUT DEFINITIONS
    //define number of bodies
    int nbodies;
    sscanf(argv[1], "%i", &nbodies);

    //define size of timestep
    float dt;
    sscanf(argv[2],"%f", &dt);
  
    //define number of iterations
    int iter;
    sscanf(argv[3],"%i", &iter);

    printf("Number of bodies: %i Number of iterations: %i\n", nbodies, iter);

    dt *= 3*pow(10, 7);

    //start timer
    float t_tot = 0.0f;

    //define memory values on host
    int bytes = nbodies*sizeof(float);
    float *x = (float*)malloc(bytes);
    float *y = (float*)malloc(bytes);
    float *z = (float*)malloc(bytes);
    float *vx = (float*)malloc(bytes);
    float *vy = (float*)malloc(bytes);
    float *vz = (float*)malloc(bytes);
    float *m = (float*)malloc(bytes);
   
    for (int i = 0; i < nbodies; ++i) {
    	x[i] = rand()/(float)RAND_MAX*2*10000 - 10000;
    	y[i] = rand()/(float)RAND_MAX*2*10000 - 10000;
    	z[i] = rand()/(float)RAND_MAX*2*10000 - 10000;
    	vx[i] = 0.f; //rand()/(float)RAND_MAX*2*1 - 1;
    	vy[i] = 0.f; //rand()/(float)RAND_MAX*2*1 - 1;
    	vz[i] = 0.f; //rand()/(float)RAND_MAX*2*1 - 1;
    	m[i] = rand()/(float)RAND_MAX*pow(10, 5);
  }
	
    fptr1 = fopen("/cosma8/data/dp004/dc-john7/snapshot_0.txt", "w");
    for(int i = 0; i<nbodies; i++){
            	fprintf(fptr1, "%f %f %f %f %f %f %f\n", x[i], y[i], z[i], vx[i], vy[i], vz[i], m[i]);}		    
    fclose(fptr1);
    

    float G = 6.67*pow(10, -11);

    //allocate device memory size
    float *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz, *d_m;
    hipMalloc(&d_x, bytes);
    hipMalloc(&d_y, bytes);
    hipMalloc(&d_z, bytes);
    hipMalloc(&d_vx, bytes);
    hipMalloc(&d_vy, bytes);
    hipMalloc(&d_vz, bytes);
    hipMalloc(&d_m, bytes);

    //copy memory to device
    hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vx, vx, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vy, vy, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vz, vz, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_m, m, bytes, hipMemcpyHostToDevice);

    for(int j =1; j<= iter; j++){
	    if(j%10==0){
		printf("Running %i \n", j);}  

	    StartTimer();	   
	    
	    //do calculation
	    force<<<dim3(256,1,1),dim3(128,1,1)>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_m, dt, nbodies, G);

	    hipError_t err = hipGetLastError();
	    if (err != hipSuccess) 
    	       printf("Error: %s\n", hipGetErrorString(err));

	    hipDeviceSynchronize();  

	    //copy memory back to host
	    hipMemcpy(x, d_x, bytes, hipMemcpyDeviceToHost);
	    hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);
   	    hipMemcpy(z, d_z, bytes, hipMemcpyDeviceToHost);
	    hipMemcpy(vx, d_vx, bytes, hipMemcpyDeviceToHost);
	    hipMemcpy(vy, d_vy, bytes, hipMemcpyDeviceToHost);
	    hipMemcpy(vz, d_vz, bytes, hipMemcpyDeviceToHost);

	    /*for(int i = 0; i<nbodies; i++){
            printf("Particle number %i- ", i);
            printf("x:%f, y:%f, z:%f, vx:%f, vy:%f, vz:%f, ", x[i], y[i], z[i], vx[i], vy[i], vz[i], m[i]);
            printf("\n");}*/

	    const float tElapsed = GetTimer();
	    t_tot += tElapsed;
	
	    char filename[100];
	    snprintf(filename, sizeof(char) * 100, "snapshot_%i.txt", j);
	
	    fptr2 = fopen(filename, "w");
	    for(int i = 0; i<nbodies; i++){
            	fprintf(fptr2, "%f %f %f %f %f %f %f\n", x[i], y[i], z[i], vx[i], vy[i], vz[i], m[i]);}
	    fclose(fptr2);


	}

	printf("Total time: %.3f ms\n", t_tot);

        /*char filename[100];
  	snprintf(filename, sizeof(char) * 32, "benchmarking-2.csv");

  	char hostname[1024];
  	hostname[1023] = '\0';
  	gethostname(hostname, 1023);
  	strtok(hostname, ".");

  	fptr3 = fopen(filename, "a");
  	fprintf(fptr3, "%s %s %i %f %i %f\n", hostname, "cuda", nbodies, dt/(3*pow(10, 7)), iter, t_tot);
  	fclose(fptr3);*/

	return 0;
}

